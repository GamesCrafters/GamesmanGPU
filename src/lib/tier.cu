#include "hip/hip_runtime.h"
#include "tier.h"
#include <iostream>
#include <sstream>
#include <stdint.h>

Tier::Tier(unsigned int num_empty_spaces, Tier *next_tier)
    : num_empty_spaces(num_empty_spaces), next_tier(next_tier) {

  // ! Use max_id >= num_positions
  num_positions = Position::max_id(num_empty_spaces) + 1;

  position_hash_to_rv = new RecursiveValue[num_positions];
  solved = false;
}

bool Tier::operator==(const Tier &other) const {
  if (num_empty_spaces != other.num_empty_spaces) {
    std::cout << "num_empty_spaces: " << num_empty_spaces;
    std::cout << " != " << other.num_empty_spaces << std::endl;
    return false;
  }
  if (num_positions != other.num_positions) {
    std::cout << "num_positions: " << num_positions;
    std::cout << " != " << other.num_positions << std::endl;
    return false;
  }
  if (solved != other.solved) {
    std::cout << "solved: " << solved;
    std::cout << " != " << other.solved << std::endl;
    return false;
  }
  // if (next_tier != other.next_tier) {
  //   return false;
  // }

  for (unsigned long long i = 0; i < num_positions; i++) {
    if (position_hash_to_rv[i] != other.position_hash_to_rv[i]) {
      std::cout << "position_hash_to_rv[" << i << "]: ";
      std::cout << position_hash_to_rv[i].format();
      std::cout << " != " << other.position_hash_to_rv[i].format() << std::endl;
      return false;
    }
  }

  return true;
}

std::string Tier::format() const {
  std::stringstream ss;
  ss << "Tier:\n";
  ss << "  num_empty_spaces: " << num_empty_spaces << "\n";
  ss << "  num_positions: " << num_positions << "\n";
  ss << "  solved: " << (solved ? "True" : "False") << "\n";
  return ss.str();
}

Tier::SolveResult Tier::solve(SolveBy solve_by) {

  auto child_position_hash_to_rv =
      next_tier ? next_tier->position_hash_to_rv : nullptr;

  auto child_num_positions = next_tier ? next_tier->num_positions : 0;

  switch (solve_by) {

  case SolveBy::CPU:
    solve_by_cpu(position_hash_to_rv, child_position_hash_to_rv,
                 num_empty_spaces, num_positions);
    break;

  case SolveBy::GPU:
    RecursiveValue *d_position_hash_to_rv, *d_child_position_hash_to_rv;

    unsigned long long position_hash_to_rv_size =
        sizeof(RecursiveValue) * num_positions;
    unsigned long long child_position_hash_to_rv_size =
        sizeof(RecursiveValue) * child_num_positions;
    hipMalloc(&d_position_hash_to_rv, position_hash_to_rv_size);
    hipMemcpy(d_position_hash_to_rv, position_hash_to_rv,
               position_hash_to_rv_size, hipMemcpyHostToDevice);
    hipMalloc(&d_child_position_hash_to_rv, child_position_hash_to_rv_size);
    hipMemcpy(d_child_position_hash_to_rv, child_position_hash_to_rv,
               child_position_hash_to_rv_size, hipMemcpyHostToDevice);

    solve_by_gpu<<<GRID_SIZE(num_positions, BLOCK_SIZE), BLOCK_SIZE>>>(
        d_position_hash_to_rv, d_child_position_hash_to_rv, num_empty_spaces,
        num_positions);

    hipMemcpy(position_hash_to_rv, d_position_hash_to_rv,
               position_hash_to_rv_size, hipMemcpyDeviceToHost);

    hipFree(d_position_hash_to_rv);
    hipFree(d_child_position_hash_to_rv);
    break;
  }

  solved = true;
  return SolveResult::Success;
}

void solve_by_cpu(RecursiveValue *position_hash_to_rv,
                  RecursiveValue *child_position_hash_to_rv,
                  unsigned int num_empty_spaces,
                  unsigned long long num_positions) {
  for (unsigned long long id = 0; id < num_positions; id++) {
    Position position = Position(id, num_empty_spaces);

    auto pv = position.primitive_value();
    if (pv != PrimitiveValue::NotPrimitive) {
      position_hash_to_rv[id] = pv.to_recursive_value();
      continue;
    }

    Position *children;
    unsigned int num_children = position.children(children);

    for (unsigned int i = 0; i < num_children; i++) {
      unsigned long long child_id = children[i].id();
      if (child_position_hash_to_rv[child_id] == RecursiveValue::Lose) {
        position_hash_to_rv[id] = RecursiveValue::Win;
        break;
      }
    }

    if (position_hash_to_rv[id] == RecursiveValue::Win) {
      delete[] children;
      continue;
    }

    for (unsigned int i = 0; i < num_children; i++) {
      unsigned long long child_id = children[i].id();
      if (child_position_hash_to_rv[child_id] == RecursiveValue::Tie) {
        position_hash_to_rv[id] = RecursiveValue::Tie;
        break;
      }
    }

    if (position_hash_to_rv[id] == RecursiveValue::Tie) {
      delete[] children;
      continue;
    }

    position_hash_to_rv[id] = RecursiveValue::Lose;
    delete[] children;
    continue;
  }
}

__global__ void solve_by_gpu(RecursiveValue *position_hash_to_rv,
                             RecursiveValue *child_position_hash_to_rv,
                             unsigned int num_empty_spaces,
                             unsigned long long num_positions) {
  unsigned long long id = (blockDim.x * blockIdx.x) + threadIdx.x;

  Position position = Position(id, num_empty_spaces);

  auto cast_rv = (uint8_t *)position_hash_to_rv;
  cast_rv[id] = 3;
}

CUDA_CALLABLE void solve_common(RecursiveValue *position_hash_to_rv,
                                RecursiveValue *child_position_hash_to_rv,
                                unsigned int num_empty_spaces,
                                unsigned long long num_positions,
                                unsigned long long id) {
  Position position = Position(id, num_empty_spaces);

  auto pv = position.primitive_value();
  if (pv != PrimitiveValue::NotPrimitive) {
    position_hash_to_rv[id] = pv.to_recursive_value();
    return;
  }

  Position *children;
  unsigned int num_children = position.children(children);

  for (unsigned int i = 0; i < num_children; i++) {
    unsigned long long child_id = children[i].id();
    if (child_position_hash_to_rv[child_id] == RecursiveValue::Lose) {
      position_hash_to_rv[id] = RecursiveValue::Win;
      break;
    }
  }

  if (position_hash_to_rv[id] == RecursiveValue::Win) {
    delete[] children;
    return;
  }

  for (unsigned int i = 0; i < num_children; i++) {
    unsigned long long child_id = children[i].id();
    if (child_position_hash_to_rv[child_id] == RecursiveValue::Tie) {
      position_hash_to_rv[id] = RecursiveValue::Tie;
      break;
    }
  }

  if (position_hash_to_rv[id] == RecursiveValue::Tie) {
    delete[] children;
    return;
  }

  position_hash_to_rv[id] = RecursiveValue::Lose;
  delete[] children;
  return;
}
