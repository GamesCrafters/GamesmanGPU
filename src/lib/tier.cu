#include "hip/hip_runtime.h"
#include "tier.h"
#include <sstream>

Tier::Tier(unsigned int num_empty_spaces, Tier *next_tier)
    : num_empty_spaces(num_empty_spaces), next_tier(next_tier) {

  // ! Use max_id instead of num_positions
  num_positions = Position::max_id(num_empty_spaces);

  position_hash_to_rv = new RecursiveValue[num_positions];
  solved = false;
}

std::string Tier::format() const {
  std::stringstream ss;
  ss << "Tier:\n";
  ss << "  num_empty_spaces: " << num_empty_spaces << "\n";
  ss << "  num_positions: " << num_positions << "\n";
  ss << "  solved: " << (solved ? "True" : "False") << "\n";
  return ss.str();
}

Tier::SolveResult Tier::solve(SolveBy solve_by) {

  auto child_position_hash_to_rv =
      next_tier ? next_tier->position_hash_to_rv : nullptr;

  auto child_num_positions = next_tier ? next_tier->num_positions : 0;

  switch (solve_by) {

  case SolveBy::CPU:
    solve_by_cpu(position_hash_to_rv, child_position_hash_to_rv);
    break;

  case SolveBy::GPU:
    RecursiveValue *d_position_hash_to_rv, *d_child_position_hash_to_rv;

    unsigned long long position_hash_to_rv_size =
        sizeof(RecursiveValue) * num_positions;
    unsigned long long child_position_hash_to_rv_size =
        sizeof(RecursiveValue) * child_num_positions;
    hipMalloc(&d_position_hash_to_rv, position_hash_to_rv_size);
    hipMemcpy(d_position_hash_to_rv, position_hash_to_rv,
               position_hash_to_rv_size, hipMemcpyHostToDevice);
    hipMalloc(&d_child_position_hash_to_rv, child_position_hash_to_rv_size);
    hipMemcpy(d_child_position_hash_to_rv, child_position_hash_to_rv,
               child_position_hash_to_rv_size, hipMemcpyHostToDevice);

    solve_by_gpu<<<GRID_SIZE(num_positions, BLOCK_SIZE), BLOCK_SIZE>>>(
        d_position_hash_to_rv, d_child_position_hash_to_rv);

    hipMemcpy(position_hash_to_rv, d_position_hash_to_rv,
               position_hash_to_rv_size, hipMemcpyDeviceToHost);

    hipFree(d_position_hash_to_rv);
    hipFree(d_child_position_hash_to_rv);
    break;
  }

  solved = true;
  return SolveResult::Success;
}

void solve_by_cpu(RecursiveValue *position_hash_to_rv,
                  RecursiveValue *child_position_hash_to_rv) {
  position_hash_to_rv[0] = RecursiveValue::Tie;
}

__global__ void solve_by_gpu(RecursiveValue *position_hash_to_rv,
                             RecursiveValue *child_position_hash_to_rv) {
  position_hash_to_rv[0] = RecursiveValue::Tie;
}
