
#include <hip/hip_runtime.h>
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  // Calculate global thread thread ID
  int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

  // Boundary check
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}
