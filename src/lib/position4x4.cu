#include "hip/hip_runtime.h"
#include <iostream>
#include <bitset>
#include <string>

#include "position4x4.h"
#include "position4x4_masks.h"

bool Position4x4::operator<(const Position4x4 &rhs) const
{
    int32_t l_rows = *(int32_t *)this->rows;
    int32_t r_rows = *(int32_t *)rhs.rows;

    int32_t l_rows_flipped = *(int32_t *)flip_along_x(this->rows).data();
    int32_t r_rows_flipped = *(int32_t *)flip_along_x(rhs.rows).data();

    int32_t l_cols = *(int32_t *)this->cols;
    int32_t r_cols = *(int32_t *)rhs.cols;

    int32_t l_cols_flipped = *(int32_t *)flip_along_x(this->cols).data();
    int32_t r_cols_flipped = *(int32_t *)flip_along_x(rhs.cols).data();

    return std::max({l_rows, l_cols,
                     l_rows_flipped, l_cols_flipped}) <
           std::max({r_rows, r_cols,
                     r_rows_flipped, r_cols_flipped});
}

bool Position4x4::has_4_in_a_row() const
{
    for (int i = 0; i < 4; i++)
    {
        if (char_has_4_in_a_row(this->rows[i]))
        {
            return true;
        }
        if (char_has_4_in_a_row(this->cols[i]))
        {
            return true;
        }
    }

    if (char_has_4_in_a_row(this->pos_diag))
    {
        return true;
    }
    if (char_has_4_in_a_row(this->neg_diag))
    {
        return true;
    }

    return false;
}

std::vector<Move> Position4x4::generate_moves() const
{
    std::vector<Move> result = std::vector<Move>();

    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            // If not taken
            if (!(this->rows[i] & TAKEN_MASKS[j]))
            {
                result.push_back(Move(Move::Piece::O, i, j));
                result.push_back(Move(Move::Piece::X, i, j));
            }
        }
    }

    return result;
}

Position4x4 Position4x4::do_move(const Move &move) const
{
    // Copy
    Position4x4 result = *this;

    // player
    result.player = Player(1 - result.player);

    // rows
    char_set_piece(result.rows[move.x], move.y, move.piece);

    // cols
    char_set_piece(result.cols[move.y], move.x, move.piece);

    // pos_diag
    if (move.x == move.y)
    {
        char_set_piece(result.pos_diag, move.x, move.piece);
    }

    // neg_diag
    if (move.x == 3 - move.y)
    {
        char_set_piece(result.neg_diag, move.x, move.piece);
    }

    // num_spaces_remain
    result.num_spaces_remain -= 1;

    return result;
}

PrimitiveValue Position4x4::primitive_value() const
{
    if (this->has_4_in_a_row())
    {
        switch (this->player)
        {
        case Player::Order:
            return PrimitiveValue::Win;
        case Player::Chaos:
            return PrimitiveValue::Lose;
        default:
            std::cerr << "Invalid player type: " << this->player << std::endl;
            throw std::invalid_argument("Invalid player type");
        }
    }

    // If no space remain
    if (!this->num_spaces_remain)
    {
        switch (this->player)
        {
        case Player::Order:
            return PrimitiveValue::Lose;
        case Player::Chaos:
            return PrimitiveValue::Win;
        default:
            std::cerr << "Invalid player type: " << this->player << std::endl;
            throw std::invalid_argument("Invalid player type");
        }
    }

    return PrimitiveValue::NotPrimitive;
}

// Formatting

std::string Position4x4::format() const
{
    std::string result = "";

    result += format_player(this->player) + "\t";
    result += format_primitive_value(this->primitive_value()) + '\t';
    result += "num_spaces_remain: " +
              std::to_string(this->num_spaces_remain) + '\n';

    for (int i = 0; i < 4; i++)
    {
        result += std::bitset<8>(this->rows[i]).to_string() + '\n';
    }

    return result;
}

std::string Position4x4::format_pretty() const
{
    std::string result = "";

    result += format_player(this->player) + "\t";
    result += format_primitive_value(this->primitive_value()) + '\t';
    result += "num_spaces_remain: " +
              std::to_string(this->num_spaces_remain) + '\n';

    for (int i = 0; i < 4; i++)
    {
        result += '|';

        char row = this->rows[i];

        for (int j = 0; j < 4; j++)
        {
            if (row & TAKEN_MASKS[3])
            {
                if (row >= (char)SET_X_3)
                {
                    result += 'X';
                }
                else
                {
                    result += 'O';
                }
            }
            else
            {
                result += ' ';
            }

            result += '|';

            row <<= 2;
        }

        result += "\n---------\n";
    }

    return result;
}
